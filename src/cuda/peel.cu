#include "hip/hip_runtime.h"
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#include <stdlib.h>

#include <hip/hip_complex.h>

#include "common.cuh"
#include "peel.h"

/**
 * Turn XYZs into UVWs. Multiple sets of XYZs over time can be converted.
 * Expects the device to be parallel over baselines.
 */
__global__ void xyzs_to_uvws_kernel(const XYZ *xyzs, const FLOAT *lmsts, UVW *uvws, RADec pointing_centre,
                                    int num_tiles, int num_baselines, int num_timesteps) {
    const int i_bl = threadIdx.x + (blockDim.x * blockIdx.x);
    if (i_bl >= num_baselines)
        return;

    const float n = (float)(num_tiles - 1);
    const float tile1f = floorf(-0.5 * sqrtf(4.0 * n * (n + 1.0) - 8.0 * i_bl + 1.0) + n + 0.5);
    const int tile2 = (int)(i_bl - tile1f * (n - (tile1f + 1.0) / 2.0) + 1.0);
    const int tile1 = (int)tile1f;

    FLOAT s_ha, c_ha, s_dec, c_dec;
    SINCOS(pointing_centre.dec, &s_dec, &c_dec);

    for (int i_time = 0; i_time < num_timesteps; i_time++) {
        XYZ xyz = xyzs[i_time * num_tiles + tile1];
        const XYZ xyz2 = xyzs[i_time * num_tiles + tile2];
        xyz.x -= xyz2.x;
        xyz.y -= xyz2.y;
        xyz.z -= xyz2.z;

        const FLOAT hour_angle = lmsts[i_time] - pointing_centre.ra;
        SINCOS(hour_angle, &s_ha, &c_ha);

        uvws[i_time * num_baselines + (int)i_bl] = UVW{
            .u = s_ha * xyz.x + c_ha * xyz.y,
            .v = -s_dec * c_ha * xyz.x + s_dec * s_ha * xyz.y + c_dec * xyz.z,
            .w = c_dec * c_ha * xyz.x - c_dec * s_ha * xyz.y + s_dec * xyz.z,
        };
    }
}

/**
 * Kernel for rotating visibilities and averaging them into "low-resolution"
 * visibilities.
 *
 * The visibilities should be ordered in time, frequency and baseline (slowest
 * to fastest). The weights should never be negative; this allows us to avoid
 * special logic when averaging.
 */
__global__ void rotate_average_kernel(const JONES *high_res_vis, const float *high_res_weights, JONES *low_res_vis,
                                      RADec pointing_centre, const int num_timesteps, const int num_tiles,
                                      const int num_baselines, const int num_freqs, const int freq_average_factor,
                                      const FLOAT *lmsts, const XYZ *xyzs, const UVW *uvws_from, UVW *uvws_to,
                                      const FLOAT *lambdas) {
    const int i_bl = threadIdx.x + (blockDim.x * blockIdx.x);
    if (i_bl >= num_baselines)
        return;

    // Prepare an "argument" for later.
    const FLOAT arg = -TAU * (uvws_to[i_bl].w - uvws_from[i_bl].w);

    for (int i_freq = 0; i_freq < num_freqs; i_freq += freq_average_factor) {
        JONES vis_weighted_sum = JONES{
            .j00_re = 0.0,
            .j00_im = 0.0,
            .j01_re = 0.0,
            .j01_im = 0.0,
            .j10_re = 0.0,
            .j10_im = 0.0,
            .j11_re = 0.0,
            .j11_im = 0.0,
        };
        FLOAT weight_sum = 0.0;

        for (int i_time = 0; i_time < num_timesteps; i_time++) {
            for (int i_freq_chunk = i_freq; i_freq_chunk < i_freq + freq_average_factor; i_freq_chunk++) {
                COMPLEX complex;
                SINCOS(arg / lambdas[i_freq_chunk], &complex.y, &complex.x);

                const int step = (i_time * num_freqs + i_freq_chunk) * num_baselines + i_bl;
                const FLOAT weight = high_res_weights[step];
                const JONES rotated_weighted_vis = high_res_vis[step] * weight * complex;

                vis_weighted_sum += rotated_weighted_vis;
                weight_sum += weight;
            }
        }

        // If `weight_sum` is bigger than 0, use it in division, otherwise just
        // divide by 1. We do this so we don't get NaN values, and we don't use
        // if statements in case the compiler optimises this better to avoid
        // warp divergence.
        vis_weighted_sum /= (weight_sum > 0.0) ? weight_sum : 1.0;

        const int low_res_step = (i_freq / freq_average_factor) * num_baselines + i_bl;
        low_res_vis[low_res_step] = vis_weighted_sum;
        // low_res_weights[low_res_step] = weight_sum;
    }
}

/**
 *
 */
__device__ void apply_iono(const JonesF32 *vis, JONES *vis_out, const FLOAT iono_const_alpha,
                           const FLOAT iono_const_beta, const int num_baselines, const int num_freqs, const UVW *uvws,
                           const FLOAT *lambdas_m) {
    const int i_bl = threadIdx.x + (blockDim.x * blockIdx.x);
    // No need to check if this thread should continue; this is a device
    // function.

    const UVW uvw = uvws[i_bl];
    const FLOAT arg = -TAU * (uvw.u * iono_const_alpha + uvw.v * iono_const_beta);

    for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
        COMPLEX complex;
        // The baseline UV is in units of metres, so we need to divide by λ to
        // use it in an exponential. But we're also multiplying by λ², so just
        // multiply by λ.
        SINCOS(arg * lambdas_m[i_freq], &complex.y, &complex.x);

        const int step = i_freq * num_baselines + i_bl;
        // TODO: Yuck
        const int step2 = i_bl * num_freqs + i_freq;
        const JonesF32 asdf = vis[step2] * complex;
        vis_out[step] = JONES{
            .j00_re = (FLOAT)asdf.j00_re,
            .j00_im = (FLOAT)asdf.j00_im,
            .j01_re = (FLOAT)asdf.j01_re,
            .j01_im = (FLOAT)asdf.j01_im,
            .j10_re = (FLOAT)asdf.j10_re,
            .j10_im = (FLOAT)asdf.j10_im,
            .j11_re = (FLOAT)asdf.j11_re,
            .j11_im = (FLOAT)asdf.j11_im,
        };
    }
}

// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// template <unsigned int blockSize> __device__ void warpReduce(volatile JonesF64 *sdata, unsigned int tid) {
__device__ void warpReduce(volatile JonesF64 *sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}
// template <unsigned int blockSize> __global__ void reduce_jones(JonesF64 *data, const int n) {
//     extern __shared__ JonesF64 sdata[];
//     unsigned int tid = threadIdx.x;
//     unsigned int i = blockIdx.x * (blockSize * 2) + tid;
//     unsigned int gridSize = blockSize * 2 * gridDim.x;
//     sdata[tid] = JonesF64{
//         .j00_re = 0.0,
//         .j00_im = 0.0,
//         .j01_re = 0.0,
//         .j01_im = 0.0,
//         .j10_re = 0.0,
//         .j10_im = 0.0,
//         .j11_re = 0.0,
//         .j11_im = 0.0,
//     };

//     while (i < n + gridSize) {
//         sdata[tid] += data[i] + data[i + blockSize];
//         i += gridSize;
//     }
//     __syncthreads();
//     if (blockSize >= 512) {
//         if (tid < 256) {
//             sdata[tid] += sdata[tid + 256];
//         }
//         __syncthreads();
//     }
//     if (blockSize >= 256) {
//         if (tid < 128) {
//             sdata[tid] += sdata[tid + 128];
//         }
//         __syncthreads();
//     }
//     if (blockSize >= 128) {
//         if (tid < 64) {
//             sdata[tid] += sdata[tid + 64];
//         }
//         __syncthreads();
//     }
//     if (tid < 32)
//         warpReduce<blockSize>(sdata, tid);
//     if (tid == 0)
//         data[blockIdx.x] = sdata[0];
//     if (tid == 0) {
//         printf("reduce %f %d\n", sdata[0].j11_im, blockIdx.x);
//     }
// }

// For each frequency, add up all of the data across baselines, dumping the
// results back into the data according to block index.
__global__ void reduce_jones(JonesF64 *data, const int num_baselines) {
    extern __shared__ JonesF64 sdata[];
    const int tid = threadIdx.x;
    const int offset = blockIdx.x * num_baselines;

    sdata[tid] = JonesF64{
        .j00_re = 0.0,
        .j00_im = 0.0,
        .j01_re = 0.0,
        .j01_im = 0.0,
        .j10_re = 0.0,
        .j10_im = 0.0,
        .j11_re = 0.0,
        .j11_im = 0.0,
    };
    int i = tid;
    while (i + blockDim.x < num_baselines) {
        sdata[tid] += data[i + offset] + data[i + offset + blockDim.x];
        i += 2 * blockDim.x;
    }
    __syncthreads();
    // Get the last few values.
    if (i < num_baselines) {
        sdata[tid] += data[i + offset];
    }
    __syncthreads();

    if (blockDim.x >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (blockDim.x >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (blockDim.x >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32) {
        warpReduce(sdata, tid);
    }
    if (tid == 0) {
        data[blockIdx.x] = sdata[0];
        // printf("reduce %f %d\n", sdata[0].j11_im, blockIdx.x);
        // printf("");
    }
}

// This is designed to take only a single block and add everything.
__global__ void reduce_jones2(JonesF64 *data, const int n, double *iono_consts) {
    extern __shared__ JonesF64 sdata[];
    int tid = threadIdx.x;

    if (tid < n) {
        sdata[tid] = data[tid];
    }
    __syncthreads();
    for (int i = n - 1; i > 0; i -= 1) {
        if (tid == i) {
            sdata[tid - 1] += sdata[tid];
        }
        __syncthreads();
    }
    if (tid == 0) {
        const double a_uu = sdata[0].j00_re;
        const double a_uv = sdata[0].j00_im;
        const double a_vv = sdata[0].j01_re;
        const double aa_u = sdata[0].j01_im;
        const double aa_v = sdata[0].j10_re;
        // const double s_vm = sdata[0].j10_im;
        // const double s_mm = sdata[0].j11_re;
        // printf("reduce2 %f\n", sdata[0].j11_im);
        const double denom = TAU * (a_uu * a_vv - a_uv * a_uv);
        iono_consts[0] += (aa_u * a_vv - aa_v * a_uv) / denom;
        iono_consts[1] += (aa_v * a_uu - aa_u * a_uv) / denom;
    }
}

/**
 * Kernel for ...
 */
__global__ void iono_loop_kernel(const JONES *vis_residual, const float *vis_weights, const JonesF32 *vis_model,
                                 JONES *vis_model_rotated, const double *iono_consts, JonesF64 *iono_fits,
                                 const int num_iterations, const int num_baselines, const int num_freqs,
                                 const FLOAT *lmsts, const UVW *uvws, const FLOAT *lambdas_m) {
    const int i_bl = threadIdx.x + (blockDim.x * blockIdx.x);
    if (i_bl >= num_baselines)
        return;

    const UVW uvw = uvws[i_bl];

    // Apply the latest iono constants to the model visibilities.
    const double iono_const_alpha = iono_consts[0];
    const double iono_const_beta = iono_consts[1];

    // TODO: Would it be better to avoid the function call?
    // TODO: Use the updated source position for the UVWs?
    apply_iono(vis_model, vis_model_rotated, iono_const_alpha, iono_const_beta, num_baselines, num_freqs, uvws,
               lambdas_m);

    for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
        const double lambda = lambdas_m[i_freq];
        const double lambda_2 = lambda * lambda;

        const double u = (double)uvw.u;
        const double v = (double)uvw.v;

        const int step = i_freq * num_baselines + i_bl;
        const double weight = vis_weights[step];
        const JONES *residual = &vis_residual[step];
        const double residual_i_re = residual->j00_re + residual->j11_re;
        const double residual_i_im = residual->j00_im + residual->j11_im;
        const JONES *model = &vis_model_rotated[step];
        const double model_i_re = model->j00_re + model->j11_re;
        const double model_i_im = model->j00_im + model->j11_im;

        const double mr = model_i_re * (residual_i_im - model_i_im);
        const double mm = model_i_re * model_i_re;

        JonesF64 j = JonesF64{
            .j00_re = lambda_2 * weight * mm * u * u,      // a_uu
            .j00_im = lambda_2 * weight * mm * u * v,      // a_uv
            .j01_re = lambda_2 * weight * mm * v * v,      // a_vv
            .j01_im = -lambda * weight * mr * u,           // aa_u
            .j10_re = -lambda * weight * mr * v,           // aa_v
            .j10_im = weight * model_i_re * residual_i_re, // s_vm
            .j11_re = weight * mm,                         // s_mm
            .j11_im = 1.0,
        };
        iono_fits[step] = j;
    }
}

__global__ void subtract_iono_kernel(JONES *vis_residual, const JonesF32 *vis_model, const double iono_const_alpha,
                                     const double iono_const_beta, const UVW *uvws, const FLOAT *lambdas_m,
                                     const int num_timesteps, const int num_baselines, const int num_freqs) {
    const int i_bl = threadIdx.x + (blockDim.x * blockIdx.x);
    if (i_bl >= num_baselines)
        return;

    for (int i_time = 0; i_time < num_timesteps; i_time++) {
        const UVW uvw = uvws[i_time * num_baselines + i_bl];
        const FLOAT arg = -TAU * (uvw.u * iono_const_alpha + uvw.v * iono_const_beta);
        for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
            const FLOAT lambda = lambdas_m[i_freq];

            COMPLEX complex;
            // The baseline UV is in units of metres, so we need to divide by λ to
            // use it in an exponential. But we're also multiplying by λ², so just
            // multiply by λ.
            SINCOS(arg * lambda, &complex.y, &complex.x);

            const int step = (i_time * num_freqs + i_freq) * num_baselines + i_bl;
            // TODO: Yuck
            const int step2 = (i_time * num_baselines + i_bl) * num_freqs + i_freq;
            JONES r = vis_residual[step];
            const JonesF32 m = vis_model[step2];

            r += m;
            r -= m * complex;
            vis_residual[step] = r;
        }
    }
}

/* Host functions */

extern "C" int xyzs_to_uvws(const XYZ *d_xyzs, const FLOAT *d_lmsts, UVW *d_uvws, RADec pointing_centre, int num_tiles,
                            int num_baselines, int num_timesteps) {
    dim3 gridDim, blockDim;
    // Thread blocks are distributed by baseline indices.
    blockDim.x = 256;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = (int)ceil((double)num_baselines / (double)blockDim.x);
    gridDim.y = 1;
    gridDim.z = 1;

    xyzs_to_uvws_kernel<<<gridDim, blockDim>>>(d_xyzs, d_lmsts, d_uvws, pointing_centre, num_tiles, num_baselines,
                                               num_timesteps);
    cudaCheck(hipPeekAtLastError());

    return 0;
}

extern "C" int rotate_average(const JONES *d_high_res_vis, const float *d_high_res_weights, JONES *d_low_res_vis,
                              RADec pointing_centre, const int num_timesteps, const int num_tiles,
                              const int num_baselines, const int num_freqs, const int freq_average_factor,
                              const FLOAT *d_lmsts, const XYZ *d_xyzs, const UVW *d_uvws_from, UVW *d_uvws_to,
                              const FLOAT *d_lambdas) {
    dim3 gridDim, blockDim;
    // Thread blocks are distributed by baseline indices.
    blockDim.x = 256;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = (int)ceil((double)num_baselines / (double)blockDim.x);
    gridDim.y = 1;
    gridDim.z = 1;

    // Prepare the "to" UVWs.
    xyzs_to_uvws_kernel<<<gridDim, blockDim>>>(d_xyzs, d_lmsts, d_uvws_to, pointing_centre, num_tiles, num_baselines,
                                               num_timesteps);
    rotate_average_kernel<<<gridDim, blockDim>>>(
        d_high_res_vis, d_high_res_weights, d_low_res_vis, pointing_centre, num_timesteps, num_tiles, num_baselines,
        num_freqs, freq_average_factor, d_lmsts, d_xyzs, d_uvws_from, d_uvws_to, d_lambdas);
    cudaCheck(hipPeekAtLastError());

    return 0;
}

extern "C" int iono_loop(const JONES *d_vis_residual, const float *d_vis_weights, const JonesF32 *d_vis_model,
                         JONES *d_vis_model_rotated, JonesF64 *d_iono_fits, double *iono_const_alpha,
                         double *iono_const_beta, const int num_timesteps, const int num_tiles, const int num_baselines,
                         const int num_freqs, const int num_iterations, const FLOAT *d_lmsts, const UVW *d_uvws,
                         const FLOAT *d_lambdas_m) {
    // Thread blocks are distributed by baseline indices.
    dim3 gridDim, blockDim;
    blockDim.x = 256;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = (int)ceil((double)num_baselines / (double)blockDim.x);
    gridDim.y = 1;
    gridDim.z = 1;
    // These are used to do ionospheric fit adding.
    dim3 gridDimAdd, blockDimAdd;
    blockDimAdd.x = 256;
    blockDimAdd.y = 1;
    blockDimAdd.z = 1;
    gridDimAdd.x = num_freqs;
    gridDimAdd.y = 1;
    gridDimAdd.z = 1;
    // And one final ionospheric fit adding.
    dim3 gridDimAdd2, blockDimAdd2;
    blockDimAdd2.x = num_freqs;
    blockDimAdd2.y = 1;
    blockDimAdd2.z = 1;
    gridDimAdd2.x = 1;
    gridDimAdd2.y = 1;
    gridDimAdd2.z = 1;

    double *d_iono_consts;
    hipMalloc(&d_iono_consts, 2 * sizeof(double));
    hipMemcpy(d_iono_consts, iono_const_alpha, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_iono_consts + 1, iono_const_beta, sizeof(double), hipMemcpyHostToDevice);

    for (int iteration = 0; iteration < num_iterations; iteration++) {
        // Do the work for one loop of the iteration.
        iono_loop_kernel<<<gridDim, blockDim>>>(d_vis_residual, d_vis_weights, d_vis_model, d_vis_model_rotated,
                                                d_iono_consts, d_iono_fits, num_tiles, num_baselines, num_freqs,
                                                d_lmsts, d_uvws, d_lambdas_m);
        cudaCheck(hipPeekAtLastError());
        // Sum the iono fits.
        reduce_jones<<<gridDimAdd, blockDimAdd, blockDimAdd.x * sizeof(JonesF64)>>>(d_iono_fits, num_baselines);
        cudaCheck(hipPeekAtLastError());
        reduce_jones2<<<gridDimAdd2, blockDimAdd2, num_freqs * sizeof(JonesF64)>>>(d_iono_fits, num_freqs,
                                                                                   d_iono_consts);
        cudaCheck(hipPeekAtLastError());

        // // Sane?
        // printf("iter %d\n", iteration);
        // hipMemcpy(iono_const_alpha, d_iono_consts, sizeof(double), hipMemcpyDeviceToHost);
        // hipMemcpy(iono_const_beta, d_iono_consts + 1, sizeof(double), hipMemcpyDeviceToHost);
        // printf("%.4e %.4e\n", *iono_const_alpha, *iono_const_beta);
    }

    hipMemcpy(iono_const_alpha, d_iono_consts, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(iono_const_beta, d_iono_consts + 1, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_iono_consts);
    // printf("%.4e %.4e\n", *iono_const_alpha, *iono_const_beta);

    return 0;
}

extern "C" int subtract_iono(JONES *d_vis_residual, const JonesF32 *d_vis_model, double iono_const_alpha,
                             double iono_const_beta, const UVW *d_uvws, const FLOAT *d_lambdas_m,
                             const int num_timesteps, const int num_baselines, const int num_freqs) {
    // Thread blocks are distributed by baseline indices.
    dim3 gridDim, blockDim;
    blockDim.x = 256;
    blockDim.y = 1;
    blockDim.z = 1;
    gridDim.x = (int)ceil((double)num_baselines / (double)blockDim.x);
    gridDim.y = 1;
    gridDim.z = 1;

    subtract_iono_kernel<<<gridDim, blockDim>>>(d_vis_residual, d_vis_model, iono_const_alpha, iono_const_beta, d_uvws,
                                                d_lambdas_m, num_timesteps, num_baselines, num_freqs);

    return 0;
}
