#include "hip/hip_runtime.h"
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#include <stdlib.h>

#include <hip/hip_complex.h>

#include "common.cuh"
#include "model.h"
#include "types.h"

const int NUM_THREADS_PER_BLOCK_POINTS = 128;
const int NUM_THREADS_PER_BLOCK_GAUSSIANS = 128;
const int NUM_THREADS_PER_BLOCK_SHAPELETS = 64;

inline __device__ JONES extrap_power_law_fd(const FLOAT freq, const JONES ref_flux_density,
                                            const FLOAT spectral_index) {
    const FLOAT flux_ratio = POW(freq / POWER_LAW_FD_REF_FREQ, spectral_index);
    return ref_flux_density * flux_ratio;
}

inline __device__ JONES extrap_curved_power_law_fd(const FLOAT freq, const JONES ref_flux_density,
                                                   const FLOAT spectral_index, const FLOAT q) {
    const FLOAT flux_ratio = POW(freq / POWER_LAW_FD_REF_FREQ, spectral_index);
    const FLOAT log_term = LOG(freq / POWER_LAW_FD_REF_FREQ);
    const FLOAT curved_component = EXP(q * log_term * log_term);
    return ref_flux_density * flux_ratio * curved_component;
}

inline __device__ FLOAT get_gaussian_envelope(const GaussianParams g_params, const UVW uvw) {
    FLOAT s_pa, c_pa;
    SINCOS(g_params.pa, &s_pa, &c_pa);
    // Temporary variables for clarity.
    const FLOAT k_x = uvw.u * s_pa + uvw.v * c_pa;
    const FLOAT k_y = uvw.u * c_pa - uvw.v * s_pa;
    return EXP(EXP_CONST * ((g_params.maj * g_params.maj) * (k_x * k_x) + (g_params.min * g_params.min) * (k_y * k_y)));
}

inline __device__ COMPLEX get_shapelet_envelope(const GaussianParams g_params, const ShapeletUV s_uv,
                                                const int num_coeffs, const ShapeletCoeff *coeffs,
                                                const FLOAT *shapelet_basis_values, const int sbf_l, const FLOAT sbf_c,
                                                const FLOAT chewie) {
    const FLOAT I_POWERS_REAL[4] = {1.0, 0.0, -1.0, 0.0};
    const FLOAT I_POWERS_IMAG[4] = {0.0, 1.0, 0.0, -1.0};

    FLOAT s_pa, c_pa;
    SINCOS(g_params.pa, &s_pa, &c_pa);

    // Temporary variables for clarity.
    const FLOAT x = s_uv.u * s_pa + s_uv.v * c_pa;
    const FLOAT y = s_uv.u * c_pa - s_uv.v * s_pa;
    const FLOAT const_x = g_params.maj * chewie;
    const FLOAT const_y = -g_params.min * chewie;
    const FLOAT x_pos = x * const_x + sbf_c;
    const FLOAT y_pos = y * const_y + sbf_c;
    int x_pos_int = (int)FLOOR(x_pos);
    int y_pos_int = (int)FLOOR(y_pos);

    COMPLEX envelope = COMPLEX{
        .x = 0.0,
        .y = 0.0,
    };
    for (int i_coeff = 0; i_coeff < num_coeffs; i_coeff++) {
        const ShapeletCoeff coeff = coeffs[i_coeff];

        FLOAT x_low = shapelet_basis_values[sbf_l * coeff.n1 + x_pos_int];
        FLOAT x_high = shapelet_basis_values[sbf_l * coeff.n1 + x_pos_int + 1];
        FLOAT u_value = x_low + (x_high - x_low) * (x_pos - FLOOR(x_pos));

        FLOAT y_low = shapelet_basis_values[sbf_l * coeff.n2 + y_pos_int];
        FLOAT y_high = shapelet_basis_values[sbf_l * coeff.n2 + y_pos_int + 1];
        FLOAT v_value = y_low + (y_high - y_low) * (y_pos - FLOOR(y_pos));

        FLOAT rest = coeff.value * u_value * v_value;

        // I_POWER_TABLE stuff. The intention is just to find the
        // appropriate power of i, i.e.:
        // index = (n1 + n2) % 4    (so that index is between 0 and 3 inclusive)
        // i^index, e.g.
        // i^0 =  1.0 + 0.0i
        // i^1 =  0.0 + 1.0i
        // i^2 = -1.0 + 0.0i
        // i^3 =  0.0 - 1.0i
        //
        // The following my attempt at doing this efficiently.
        int i_power_index = (coeff.n1 + coeff.n2) % 4;
        COMPLEX i_power = COMPLEX{
            .x = I_POWERS_REAL[i_power_index],
            .y = I_POWERS_IMAG[i_power_index],
        };

        envelope += i_power * rest;
    }

    return envelope;
}

__global__ void model_points_fee_kernel_small_source_count(const int num_freqs, const int num_baselines,
                                                           const FLOAT *freqs, const UVW *uvws, const Points comps,
                                                           const JONES *beam_jones, const int *tile_map,
                                                           const int *freq_map, int num_fee_freqs,
                                                           const int *tile_index_to_unflagged_tile_index_map,
                                                           JonesF32 *vis) {
    // The 0-indexed number of tiles as a float.
    const float num_tiles = (sqrtf(1.0f + 8.0f * (float)num_baselines) - 1.0f) / 2.0f;
    const int num_directions = comps.num_power_laws + comps.num_curved_power_laws + comps.num_lists;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_baselines * num_freqs; i += gridDim.x * blockDim.x) {
        const int i_bl = i % num_baselines;
        const int i_freq = i / num_baselines;

        // Get tile indices for this baseline to get the correct beam responses.
        const float tile1f =
            floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
        const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
        const int i_tile1 = (int)tile1f;

        // `i_j1_row` and `i_j2_row` are indices into beam responses.
        const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
        const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

        const FLOAT freq = freqs[i_freq];
        const UVW uvw = uvws[i_bl] * freq / VEL_C;

        const int i_col = freq_map[i_freq];
        const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col);
        const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col);

        COMPLEX geom;
        JONES delta_vis = JONES{
            .j00_re = 0.0,
            .j00_im = 0.0,
            .j01_re = 0.0,
            .j01_im = 0.0,
            .j10_re = 0.0,
            .j10_im = 0.0,
            .j11_re = 0.0,
            .j11_im = 0.0,
        };

        for (int i_comp = 0; i_comp < comps.num_power_laws; i_comp++) {
            // Estimate a flux density from the reference FD and spectral index.
            JONES fd = extrap_power_law_fd(freq, comps.power_law_fds[i_comp], comps.power_law_sis[i_comp]);
            apply_beam(j1++, &fd, j2++);

            // Measurement equation. The 2 PI is already multiplied on the LMN
            // terms (as well as a -1 on the n).
            const LmnRime lmn = comps.power_law_lmns[i_comp];
            SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &geom.y, &geom.x);
            delta_vis += fd * geom;
        }

        for (int i_comp = 0; i_comp < comps.num_curved_power_laws; i_comp++) {
            JONES fd =
                extrap_curved_power_law_fd(freq, comps.curved_power_law_fds[i_comp], comps.curved_power_law_sis[i_comp],
                                           comps.curved_power_law_qs[i_comp]);
            apply_beam(j1++, &fd, j2++);

            const LmnRime lmn = comps.curved_power_law_lmns[i_comp];
            SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &geom.y, &geom.x);
            delta_vis += fd * geom;
        }

        for (int i_comp = 0; i_comp < comps.num_lists; i_comp++) {
            JONES fd = comps.list_fds[i_freq * comps.num_lists + i_comp];
            apply_beam(j1++, &fd, j2++);

            const LmnRime lmn = comps.list_lmns[i_comp];
            SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &geom.y, &geom.x);
            delta_vis += fd * geom;
        }

        // Visibilities are ordered over baselines and frequencies, with
        // baselines moving faster than frequencies.
        vis[i_freq * num_baselines + i_bl] += delta_vis;
    }
}

/**
 * Kernel for calculating point-source-component visibilities attenuated by the
 * FEE beam.
 */
__global__ void model_points_fee_kernel_large_source_count(const int num_freqs, const int num_baselines,
                                                           const FLOAT *freqs, const UVW *uvws, const Points comps,
                                                           const JONES *beam_jones, const int *tile_map,
                                                           const int *freq_map, int num_fee_freqs,
                                                           const int *tile_index_to_unflagged_tile_index_map,
                                                           JonesF32 *vis) {
    // Set up shared memory for later, to cache global memory (access global
    // memory coalesced, rather than element by element).
    __shared__ JONES s_fds[NUM_THREADS_PER_BLOCK_POINTS];
    __shared__ FLOAT s_sis[NUM_THREADS_PER_BLOCK_POINTS];
    __shared__ FLOAT s_qs[NUM_THREADS_PER_BLOCK_POINTS];
    __shared__ LmnRime s_lmns[NUM_THREADS_PER_BLOCK_POINTS];

    // Total number of threads across all thread blocks.
    const int total_num_threads = gridDim.x * blockDim.x;
    // The 0-indexed number of tiles as a float.
    const float num_tiles = (sqrtf(1.0f + 8.0f * (float)num_baselines) - 1.0f) / 2.0f;
    const int num_directions = comps.num_power_laws + comps.num_curved_power_laws + comps.num_lists;

    // Power-law components.
    for (int i_comp_chunk = 0; i_comp_chunk < comps.num_power_laws; i_comp_chunk += NUM_THREADS_PER_BLOCK_POINTS) {
        // Populate the shared memory.
        if (i_comp_chunk + threadIdx.x < comps.num_power_laws) {
            s_fds[threadIdx.x] = comps.power_law_fds[i_comp_chunk + threadIdx.x];
            s_sis[threadIdx.x] = comps.power_law_sis[i_comp_chunk + threadIdx.x];
            s_lmns[threadIdx.x] = comps.power_law_lmns[i_comp_chunk + threadIdx.x];
        }
        __syncthreads();

        // Baseline index.
        for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
            const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

            // Get tile indices for this baseline to get the correct beam responses.
            const float tile1f =
                floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
            const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
            const int i_tile1 = (int)tile1f;

            // `i_j1_row` and `i_j2_row` are indices into beam responses.
            const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
            const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

            for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
                const FLOAT freq = freqs[i_freq];
                const UVW uvw = uvw_on_hz * freq;

                const int i_col = freq_map[i_freq];
                const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col) + i_comp_chunk;
                const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col) + i_comp_chunk;

                COMPLEX geom;
                JONES delta_vis = JONES{
                    .j00_re = 0.0,
                    .j00_im = 0.0,
                    .j01_re = 0.0,
                    .j01_im = 0.0,
                    .j10_re = 0.0,
                    .j10_im = 0.0,
                    .j11_re = 0.0,
                    .j11_im = 0.0,
                };

                for (int i_comp = 0;
                     i_comp + i_comp_chunk < comps.num_power_laws && i_comp < NUM_THREADS_PER_BLOCK_POINTS; i_comp++) {
                    // Estimate a flux density from the reference FD and spectral
                    // index.
                    JONES fd = extrap_power_law_fd(freq, s_fds[i_comp], s_sis[i_comp]);
                    apply_beam(j1++, &fd, j2++);

                    // Measurement equation. The 2 PI is already multiplied on the
                    // LMN terms (as well as a -1 on the n).
                    const LmnRime lmn = s_lmns[i_comp];
                    SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &geom.y, &geom.x);
                    delta_vis += fd * geom;
                }

                // Visibilities are ordered over baselines and frequencies, with
                // baselines moving faster than frequencies.
                vis[i_freq * num_baselines + i_bl] += delta_vis;
            }
        }
        __syncthreads();
    }

    // Curved-power-law components.
    for (int i_comp_chunk = 0; i_comp_chunk < comps.num_curved_power_laws;
         i_comp_chunk += NUM_THREADS_PER_BLOCK_POINTS) {
        // Populate the shared memory.
        if (i_comp_chunk + threadIdx.x < comps.num_curved_power_laws) {
            s_fds[threadIdx.x] = comps.curved_power_law_fds[i_comp_chunk + threadIdx.x];
            s_sis[threadIdx.x] = comps.curved_power_law_sis[i_comp_chunk + threadIdx.x];
            s_qs[threadIdx.x] = comps.curved_power_law_qs[i_comp_chunk + threadIdx.x];
            s_lmns[threadIdx.x] = comps.curved_power_law_lmns[i_comp_chunk + threadIdx.x];
        }
        __syncthreads();

        for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
            const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

            // Get tile indices for this baseline to get the correct beam responses.
            const float tile1f =
                floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
            const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
            const int i_tile1 = (int)tile1f;

            // `i_j1_row` and `i_j2_row` are indices into beam responses.
            const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
            const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

            for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
                const FLOAT freq = freqs[i_freq];
                const UVW uvw = uvw_on_hz * freq;

                const int i_col = freq_map[i_freq];
                const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col) + i_comp_chunk +
                                  comps.num_power_laws;
                const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col) + i_comp_chunk +
                                  comps.num_power_laws;

                COMPLEX geom;
                JONES delta_vis = JONES{
                    .j00_re = 0.0,
                    .j00_im = 0.0,
                    .j01_re = 0.0,
                    .j01_im = 0.0,
                    .j10_re = 0.0,
                    .j10_im = 0.0,
                    .j11_re = 0.0,
                    .j11_im = 0.0,
                };

                for (int i_comp = 0;
                     i_comp + i_comp_chunk < comps.num_curved_power_laws && i_comp < NUM_THREADS_PER_BLOCK_POINTS;
                     i_comp++) {
                    JONES fd = extrap_curved_power_law_fd(freq, s_fds[i_comp], s_sis[i_comp], s_qs[i_comp]);
                    apply_beam(j1++, &fd, j2++);

                    const LmnRime lmn = s_lmns[i_comp];
                    SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &geom.y, &geom.x);
                    delta_vis += fd * geom;
                }

                vis[i_freq * num_baselines + i_bl] += delta_vis;
            }
        }
        __syncthreads();
    }

    // List components.
    __shared__ short s_i_freqs[NUM_THREADS_PER_BLOCK_POINTS];
    for (int i_comp_chunk = 0; i_comp_chunk < comps.num_lists * num_freqs;
         i_comp_chunk += NUM_THREADS_PER_BLOCK_POINTS) {
        // Populate the shared memory.
        if (i_comp_chunk + threadIdx.x < comps.num_lists * num_freqs) {
            s_fds[threadIdx.x] = comps.list_fds[i_comp_chunk + threadIdx.x];
            const int i_comp = (i_comp_chunk + threadIdx.x) % comps.num_lists;
            s_lmns[threadIdx.x] = comps.list_lmns[i_comp];
            s_i_freqs[threadIdx.x] = (i_comp_chunk + threadIdx.x) / comps.num_lists;
        } else {
            s_i_freqs[threadIdx.x] = -1;
        }
        __syncthreads();

        for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
            const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

            // Get tile indices for this baseline to get the correct beam responses.
            const float tile1f =
                floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
            const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
            const int i_tile1 = (int)tile1f;

            // `i_j1_row` and `i_j2_row` are indices into beam responses.
            const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
            const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

            COMPLEX geom;
            JONES delta_vis = JONES{
                .j00_re = 0.0,
                .j00_im = 0.0,
                .j01_re = 0.0,
                .j01_im = 0.0,
                .j10_re = 0.0,
                .j10_im = 0.0,
                .j11_re = 0.0,
                .j11_im = 0.0,
            };

            for (int i = 0; i + i_comp_chunk < comps.num_lists * num_freqs && i < NUM_THREADS_PER_BLOCK_POINTS; i++) {
                const int i_freq = s_i_freqs[i];
                const FLOAT freq = freqs[i_freq];
                const UVW uvw = uvw_on_hz * freq;

                // Interestingly, caching the modulus result below in shared
                // memory makes the code slower than just doing it here.
                const int i_comp = (i + i_comp_chunk) % comps.num_lists;
                const int i_col = freq_map[i_freq];
                const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col) + i_comp +
                                  comps.num_power_laws + comps.num_curved_power_laws;
                const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col) + i_comp +
                                  comps.num_power_laws + comps.num_curved_power_laws;

                // Get the flux density for this frequency.
                JONES fd = s_fds[i];
                apply_beam(j1, &fd, j2);

                const LmnRime lmn = s_lmns[i];
                SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &geom.y, &geom.x);
                delta_vis += fd * geom;

                // Write the results if we're done with this frequency.
                if (i + 1 == NUM_THREADS_PER_BLOCK_POINTS || s_i_freqs[i + 1] != i_freq) {
                    vis[i_freq * num_baselines + i_bl] += delta_vis;
                    delta_vis = JONES{
                        .j00_re = 0.0,
                        .j00_im = 0.0,
                        .j01_re = 0.0,
                        .j01_im = 0.0,
                        .j10_re = 0.0,
                        .j10_im = 0.0,
                        .j11_re = 0.0,
                        .j11_im = 0.0,
                    };
                }
            }
        }
        __syncthreads();
    }
}

/**
 * Kernel for calculating Gaussian-source-component visibilities.
 */
__global__ void
model_gaussians_fee_kernel_large_source_count(const int num_freqs, const int num_baselines, const FLOAT *freqs,
                                              const UVW *uvws, const Gaussians comps, const JONES *beam_jones,
                                              const int *tile_map, const int *freq_map, const int num_fee_freqs,
                                              const int *tile_index_to_unflagged_tile_index_map, JonesF32 *vis) {
    // Set up shared memory for later, to cache global memory (access global
    // memory coalesced, rather than element by element).
    __shared__ JONES s_fds[NUM_THREADS_PER_BLOCK_GAUSSIANS];
    __shared__ FLOAT s_sis[NUM_THREADS_PER_BLOCK_GAUSSIANS];
    __shared__ FLOAT s_qs[NUM_THREADS_PER_BLOCK_GAUSSIANS];
    __shared__ LmnRime s_lmns[NUM_THREADS_PER_BLOCK_GAUSSIANS];
    __shared__ GaussianParams s_gps[NUM_THREADS_PER_BLOCK_GAUSSIANS];

    // Total number of threads across all thread blocks.
    const int total_num_threads = gridDim.x * blockDim.x;
    // The 0-indexed number of tiles as a float.
    const float num_tiles = (sqrtf(1.0f + 8.0f * (float)num_baselines) - 1.0f) / 2.0f;
    const int num_directions = comps.num_power_laws + comps.num_curved_power_laws + comps.num_lists;

    // Power-law components.
    for (int i_comp_chunk = 0; i_comp_chunk < comps.num_power_laws; i_comp_chunk += NUM_THREADS_PER_BLOCK_GAUSSIANS) {
        // Populate the shared memory.
        if (i_comp_chunk + threadIdx.x < comps.num_power_laws) {
            s_fds[threadIdx.x] = comps.power_law_fds[i_comp_chunk + threadIdx.x];
            s_sis[threadIdx.x] = comps.power_law_sis[i_comp_chunk + threadIdx.x];
            s_lmns[threadIdx.x] = comps.power_law_lmns[i_comp_chunk + threadIdx.x];
            s_gps[threadIdx.x] = comps.power_law_gps[i_comp_chunk + threadIdx.x];
        }
        __syncthreads();

        // Baseline index.
        for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
            const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

            // Get tile indices for this baseline to get the correct beam responses.
            const float tile1f =
                floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
            const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
            const int i_tile1 = (int)tile1f;

            // `i_j1_row` and `i_j2_row` are indices into beam responses.
            const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
            const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

            for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
                const FLOAT freq = freqs[i_freq];
                const UVW uvw = uvw_on_hz * freq;

                const int i_col = freq_map[i_freq];
                const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col) + i_comp_chunk;
                const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col) + i_comp_chunk;

                COMPLEX complex;
                JONES delta_vis = JONES{
                    .j00_re = 0.0,
                    .j00_im = 0.0,
                    .j01_re = 0.0,
                    .j01_im = 0.0,
                    .j10_re = 0.0,
                    .j10_im = 0.0,
                    .j11_re = 0.0,
                    .j11_im = 0.0,
                };

                for (int i_comp = 0;
                     i_comp + i_comp_chunk < comps.num_power_laws && i_comp < NUM_THREADS_PER_BLOCK_GAUSSIANS;
                     i_comp++) {
                    // Estimate a flux density from the reference FD and spectral
                    // index.
                    JONES fd = extrap_power_law_fd(freq, s_fds[i_comp], s_sis[i_comp]);
                    apply_beam(j1++, &fd, j2++);

                    // Measurement equation. The 2 PI is already multiplied on the
                    // LMN terms (as well as a -1 on the n).
                    const LmnRime lmn = s_lmns[i_comp];
                    SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

                    // Scale by envelope.
                    const FLOAT envelope = get_gaussian_envelope(s_gps[i_comp], uvw);
                    delta_vis += fd * complex * envelope;
                }

                // Visibilities are ordered over baselines and frequencies, with
                // baselines moving faster than frequencies.
                vis[i_freq * num_baselines + i_bl] += delta_vis;
            }
        }
        __syncthreads();
    }

    // Curved-power-law components.
    for (int i_comp_chunk = 0; i_comp_chunk < comps.num_curved_power_laws;
         i_comp_chunk += NUM_THREADS_PER_BLOCK_GAUSSIANS) {
        // Populate the shared memory.
        if (i_comp_chunk + threadIdx.x < comps.num_curved_power_laws) {
            s_fds[threadIdx.x] = comps.curved_power_law_fds[i_comp_chunk + threadIdx.x];
            s_sis[threadIdx.x] = comps.curved_power_law_sis[i_comp_chunk + threadIdx.x];
            s_qs[threadIdx.x] = comps.curved_power_law_qs[i_comp_chunk + threadIdx.x];
            s_lmns[threadIdx.x] = comps.curved_power_law_lmns[i_comp_chunk + threadIdx.x];
            s_gps[threadIdx.x] = comps.curved_power_law_gps[i_comp_chunk + threadIdx.x];
        }
        __syncthreads();

        for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
            const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

            // Get tile indices for this baseline to get the correct beam responses.
            const float tile1f =
                floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
            const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
            const int i_tile1 = (int)tile1f;

            // `i_j1_row` and `i_j2_row` are indices into beam responses.
            const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
            const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

            for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
                const FLOAT freq = freqs[i_freq];
                const UVW uvw = uvw_on_hz * freq;

                const int i_col = freq_map[i_freq];
                const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col) + i_comp_chunk +
                                  comps.num_power_laws;
                const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col) + i_comp_chunk +
                                  comps.num_power_laws;

                COMPLEX complex;
                JONES delta_vis = JONES{
                    .j00_re = 0.0,
                    .j00_im = 0.0,
                    .j01_re = 0.0,
                    .j01_im = 0.0,
                    .j10_re = 0.0,
                    .j10_im = 0.0,
                    .j11_re = 0.0,
                    .j11_im = 0.0,
                };

                for (int i_comp = 0;
                     i_comp + i_comp_chunk < comps.num_curved_power_laws && i_comp < NUM_THREADS_PER_BLOCK_GAUSSIANS;
                     i_comp++) {
                    JONES fd = extrap_curved_power_law_fd(freq, s_fds[i_comp], s_sis[i_comp], s_qs[i_comp]);
                    apply_beam(j1++, &fd, j2++);

                    const LmnRime lmn = s_lmns[i_comp];
                    SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

                    const FLOAT envelope = get_gaussian_envelope(s_gps[i_comp], uvw);
                    delta_vis += fd * complex * envelope;
                }

                vis[i_freq * num_baselines + i_bl] += delta_vis;
            }
        }
        __syncthreads();
    }

    // List components.
    __shared__ short s_i_freqs[NUM_THREADS_PER_BLOCK_GAUSSIANS];
    for (int i_comp_chunk = 0; i_comp_chunk < comps.num_lists * num_freqs;
         i_comp_chunk += NUM_THREADS_PER_BLOCK_GAUSSIANS) {
        // Populate the shared memory.
        if (i_comp_chunk + threadIdx.x < comps.num_lists * num_freqs) {
            s_fds[threadIdx.x] = comps.list_fds[i_comp_chunk + threadIdx.x];
            const int i_comp = (i_comp_chunk + threadIdx.x) % comps.num_lists;
            s_lmns[threadIdx.x] = comps.list_lmns[i_comp];
            s_gps[threadIdx.x] = comps.list_gps[i_comp];
            s_i_freqs[threadIdx.x] = (i_comp_chunk + threadIdx.x) / comps.num_lists;
        } else {
            s_i_freqs[threadIdx.x] = -1;
        }
        __syncthreads();

        for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
            const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

            // Get tile indices for this baseline to get the correct beam responses.
            const float tile1f =
                floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
            const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
            const int i_tile1 = (int)tile1f;

            // `i_j1_row` and `i_j2_row` are indices into beam responses.
            const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
            const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

            COMPLEX complex;
            JONES delta_vis = JONES{
                .j00_re = 0.0,
                .j00_im = 0.0,
                .j01_re = 0.0,
                .j01_im = 0.0,
                .j10_re = 0.0,
                .j10_im = 0.0,
                .j11_re = 0.0,
                .j11_im = 0.0,
            };

            for (int i = 0; i + i_comp_chunk < comps.num_lists * num_freqs && i < NUM_THREADS_PER_BLOCK_GAUSSIANS;
                 i++) {
                const int i_freq = s_i_freqs[i];
                const FLOAT freq = freqs[i_freq];
                const UVW uvw = uvw_on_hz * freq;

                const int i_comp = (i + i_comp_chunk) % comps.num_lists;
                const int i_col = freq_map[i_freq];
                const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col) + i_comp +
                                  comps.num_power_laws + comps.num_curved_power_laws;
                const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col) + i_comp +
                                  comps.num_power_laws + comps.num_curved_power_laws;

                // Get the flux density for this frequency.
                JONES fd = s_fds[i];
                apply_beam(j1, &fd, j2);

                const LmnRime lmn = s_lmns[i];
                SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

                const FLOAT envelope = get_gaussian_envelope(s_gps[i], uvw);
                delta_vis += fd * complex * envelope;

                // Write the results if we're done with this frequency.
                if (i + 1 == NUM_THREADS_PER_BLOCK_GAUSSIANS || s_i_freqs[i + 1] != i_freq) {
                    vis[i_freq * num_baselines + i_bl] += delta_vis;
                    delta_vis = JONES{
                        .j00_re = 0.0,
                        .j00_im = 0.0,
                        .j01_re = 0.0,
                        .j01_im = 0.0,
                        .j10_re = 0.0,
                        .j10_im = 0.0,
                        .j11_re = 0.0,
                        .j11_im = 0.0,
                    };
                }
            }
        }
        __syncthreads();
    }
}

/**
 * Kernel for calculating shapelet-source-component visibilities.
 *
 * `*_shapelet_coeffs` is actually a flattened array-of-arrays. The size of each
 * sub-array is given by an element of `*_num_shapelet_coeffs`.
 */
__global__ void model_shapelets_fee_kernel_large_source_count(
    const int num_freqs, const int num_baselines, const FLOAT *freqs, const UVW *uvws, const Shapelets comps,
    const FLOAT *shapelet_basis_values, const int sbf_l, const FLOAT sbf_c, const FLOAT sbf_dx, const JONES *beam_jones,
    const int *tile_map, const int *freq_map, const int num_fee_freqs,
    const int *tile_index_to_unflagged_tile_index_map, JonesF32 *vis) {
    // // Set up shared memory for later, to cache global memory (access global
    // // memory coalesced, rather than element by element).
    // __shared__ JONES s_fds[NUM_THREADS_PER_BLOCK_SHAPELETS];
    // __shared__ FLOAT s_sis[NUM_THREADS_PER_BLOCK_SHAPELETS];
    // __shared__ FLOAT s_qs[NUM_THREADS_PER_BLOCK_SHAPELETS];
    // __shared__ LmnRime s_lmns[NUM_THREADS_PER_BLOCK_SHAPELETS];
    // __shared__ GaussianParams s_gps[NUM_THREADS_PER_BLOCK_SHAPELETS];
    // __shared__ int s_num_coeffs[NUM_THREADS_PER_BLOCK_SHAPELETS];

    // Baseline index.
    const int i_bl = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_bl >= num_baselines)
        return;

    // // Total number of threads across all thread blocks.
    // const int total_num_threads = gridDim.x * blockDim.x;
    // The 0-indexed number of tiles as a float.
    const float num_tiles = (sqrtf(1.0f + 8.0f * (float)num_baselines) - 1.0f) / 2.0f;
    const int num_directions = comps.num_power_laws + comps.num_curved_power_laws + comps.num_lists;
    // Shapelet-specific constant.
    const FLOAT chewie = SQRT_FRAC_PI_SQ_2_LN_2 / sbf_dx;

    // Get tile indices for this baseline to get the correct beam responses.
    const float tile1f =
        floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
    const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
    const int i_tile1 = (int)tile1f;

    // `i_j1_row` and `i_j2_row` are indices into beam responses.
    const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
    const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

    for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
        const FLOAT freq = freqs[i_freq];
        const FLOAT one_on_lambda = freq / VEL_C;
        const UVW uvw = uvws[i_bl] * one_on_lambda;

        const int i_col = freq_map[i_freq];
        const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col);
        const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col);

        COMPLEX complex;
        JONES delta_vis = JONES{
            .j00_re = 0.0,
            .j00_im = 0.0,
            .j01_re = 0.0,
            .j01_im = 0.0,
            .j10_re = 0.0,
            .j10_im = 0.0,
            .j11_re = 0.0,
            .j11_im = 0.0,
        };

        const ShapeletCoeff *shapelet_coeffs = comps.power_law_shapelet_coeffs;
        for (int i_comp = 0; i_comp < comps.num_power_laws; i_comp++) {
            JONES fd = extrap_power_law_fd(freq, comps.power_law_fds[i_comp], comps.power_law_sis[i_comp]);
            apply_beam(j1++, &fd, j2++);

            const LmnRime lmn = comps.power_law_lmns[i_comp];
            SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

            // Scale by envelope.
            ShapeletUV s_uv = comps.power_law_shapelet_uvs[i_bl * comps.num_power_laws + i_comp] * one_on_lambda;
            int num_coeffs = comps.power_law_num_shapelet_coeffs[i_comp];
            COMPLEX envelope = get_shapelet_envelope(comps.power_law_gps[i_comp], s_uv, num_coeffs, shapelet_coeffs,
                                                     shapelet_basis_values, sbf_l, sbf_c, chewie);
            shapelet_coeffs += num_coeffs;

            delta_vis += fd * complex * envelope;
        }

        shapelet_coeffs = comps.curved_power_law_shapelet_coeffs;
        for (int i_comp = 0; i_comp < comps.num_curved_power_laws; i_comp++) {
            JONES fd =
                extrap_curved_power_law_fd(freq, comps.curved_power_law_fds[i_comp], comps.curved_power_law_sis[i_comp],
                                           comps.curved_power_law_qs[i_comp]);
            apply_beam(j1++, &fd, j2++);

            const LmnRime lmn = comps.curved_power_law_lmns[i_comp];
            SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

            ShapeletUV s_uv =
                comps.curved_power_law_shapelet_uvs[i_bl * comps.num_curved_power_laws + i_comp] * one_on_lambda;
            int num_coeffs = comps.curved_power_law_num_shapelet_coeffs[i_comp];
            COMPLEX envelope = get_shapelet_envelope(comps.curved_power_law_gps[i_comp], s_uv, num_coeffs,
                                                     shapelet_coeffs, shapelet_basis_values, sbf_l, sbf_c, chewie);
            shapelet_coeffs += num_coeffs;

            delta_vis += fd * complex * envelope;
        }

        shapelet_coeffs = comps.list_shapelet_coeffs;
        for (int i_comp = 0; i_comp < comps.num_lists; i_comp++) {
            JONES fd = comps.list_fds[i_freq * comps.num_lists + i_comp];
            apply_beam(j1++, &fd, j2++);

            const LmnRime lmn = comps.list_lmns[i_comp];
            SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

            ShapeletUV s_uv = comps.list_shapelet_uvs[i_bl * comps.num_lists + i_comp] * one_on_lambda;
            int num_coeffs = comps.list_num_shapelet_coeffs[i_comp];
            COMPLEX envelope = get_shapelet_envelope(comps.list_gps[i_comp], s_uv, num_coeffs, shapelet_coeffs,
                                                     shapelet_basis_values, sbf_l, sbf_c, chewie);
            shapelet_coeffs += num_coeffs;

            delta_vis += fd * complex * envelope;
        }

        vis[i_freq * num_baselines + i_bl] += delta_vis;
    }

    // // Power-law components.
    // for (int i_comp_chunk = 0; i_comp_chunk < comps.num_power_laws; i_comp_chunk += NUM_THREADS_PER_BLOCK_SHAPELETS)
    // {
    //     // Populate the shared memory.
    //     if (i_comp_chunk + threadIdx.x < comps.num_power_laws) {
    //         s_fds[threadIdx.x] = comps.power_law_fds[i_comp_chunk + threadIdx.x];
    //         s_sis[threadIdx.x] = comps.power_law_sis[i_comp_chunk + threadIdx.x];
    //         s_lmns[threadIdx.x] = comps.power_law_lmns[i_comp_chunk + threadIdx.x];
    //         s_gps[threadIdx.x] = comps.power_law_gps[i_comp_chunk + threadIdx.x];
    //         s_num_coeffs[threadIdx.x] = comps.power_law_num_shapelet_coeffs[i_comp_chunk + threadIdx.x];
    //     }
    //     __syncthreads();

    //     for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
    //         const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

    //         // Get tile indices for this baseline to get the correct beam responses.
    //         const float tile1f =
    //             floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
    //         const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
    //         const int i_tile1 = (int)tile1f;

    //         // `i_j1_row` and `i_j2_row` are indices into beam responses.
    //         const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
    //         const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

    //         ShapeletUV *s_uvs = comps.power_law_shapelet_uvs + i_bl * comps.num_power_laws + i_comp_chunk;

    //         for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
    //             const FLOAT freq = freqs[i_freq];
    //             const UVW uvw = uvw_on_hz * freq;
    //             const FLOAT one_on_lambda = freq / VEL_C;

    //             const int i_col = freq_map[i_freq];
    //             const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col);
    //             const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col);

    //             COMPLEX complex;
    //             JONES delta_vis = JONES{
    //                 .j00_re = 0.0,
    //                 .j00_im = 0.0,
    //                 .j01_re = 0.0,
    //                 .j01_im = 0.0,
    //                 .j10_re = 0.0,
    //                 .j10_im = 0.0,
    //                 .j11_re = 0.0,
    //                 .j11_im = 0.0,
    //             };

    //             for (int i_comp = 0;
    //                  i_comp + i_comp_chunk < comps.num_power_laws && i_comp < NUM_THREADS_PER_BLOCK_SHAPELETS;
    //                  i_comp++) {
    //                 // Estimate a flux density from the reference FD and spectral
    //                 // index.
    //                 JONES fd = extrap_power_law_fd(freq, s_fds[i_comp], s_sis[i_comp]);
    //                 apply_beam(j1++, &fd, j2++);

    //                 // Measurement equation. The 2 PI is already multiplied on the
    //                 // LMN terms (as well as a -1 on the n).
    //                 const LmnRime lmn = s_lmns[i_comp];
    //                 SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

    //                 // Scale by envelope.
    //                 const ShapeletUV s_uv = s_uvs[i_comp] * one_on_lambda;
    //                 const COMPLEX envelope = get_shapelet_envelope(s_gps[i_comp], s_uv, s_num_coeffs[i_comp],
    //                                                                comps.power_law_shapelet_coeffs,
    //                                                                shapelet_basis_values, sbf_l, sbf_c, chewie);

    //                 delta_vis += fd * complex * envelope;
    //             }

    //             // Visibilities are ordered over baselines and frequencies, with
    //             // baselines moving faster than frequencies.
    //             vis[i_freq * num_baselines + i_bl] += delta_vis;
    //         }
    //     }
    //     __syncthreads();
    // }

    // // Curved-power-law components.
    // for (int i_comp_chunk = 0; i_comp_chunk < comps.num_curved_power_laws;
    //      i_comp_chunk += NUM_THREADS_PER_BLOCK_SHAPELETS) {
    //     // Populate the shared memory.
    //     if (i_comp_chunk + threadIdx.x < comps.num_curved_power_laws) {
    //         s_fds[threadIdx.x] = comps.curved_power_law_fds[i_comp_chunk + threadIdx.x];
    //         s_sis[threadIdx.x] = comps.curved_power_law_sis[i_comp_chunk + threadIdx.x];
    //         s_qs[threadIdx.x] = comps.curved_power_law_qs[i_comp_chunk + threadIdx.x];
    //         s_lmns[threadIdx.x] = comps.curved_power_law_lmns[i_comp_chunk + threadIdx.x];
    //         s_gps[threadIdx.x] = comps.curved_power_law_gps[i_comp_chunk + threadIdx.x];
    //         s_num_coeffs[threadIdx.x] = comps.curved_power_law_num_shapelet_coeffs[i_comp_chunk + threadIdx.x];
    //     }
    //     __syncthreads();

    //     for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
    //         const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

    //         // Get tile indices for this baseline to get the correct beam responses.
    //         const float tile1f =
    //             floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
    //         const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
    //         const int i_tile1 = (int)tile1f;

    //         // `i_j1_row` and `i_j2_row` are indices into beam responses.
    //         const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
    //         const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

    //         ShapeletUV *s_uvs = comps.curved_power_law_shapelet_uvs + i_bl * comps.num_curved_power_laws;

    //         for (int i_freq = 0; i_freq < num_freqs; i_freq++) {
    //             const FLOAT freq = freqs[i_freq];
    //             const UVW uvw = uvw_on_hz * freq;
    //             const FLOAT one_on_lambda = freq / VEL_C;

    //             const int i_col = freq_map[i_freq];
    //             const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col);
    //             const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col);

    //             COMPLEX complex;
    //             JONES delta_vis = JONES{
    //                 .j00_re = 0.0,
    //                 .j00_im = 0.0,
    //                 .j01_re = 0.0,
    //                 .j01_im = 0.0,
    //                 .j10_re = 0.0,
    //                 .j10_im = 0.0,
    //                 .j11_re = 0.0,
    //                 .j11_im = 0.0,
    //             };

    //             for (int i_comp = 0;
    //                  i_comp + i_comp_chunk < comps.num_curved_power_laws && i_comp < NUM_THREADS_PER_BLOCK_SHAPELETS;
    //                  i_comp++) {
    //                 // Estimate a flux density from the reference FD and spectral
    //                 // index.
    //                 JONES fd = extrap_curved_power_law_fd(freq, s_fds[i_comp], s_sis[i_comp], s_qs[i_comp]);
    //                 apply_beam(j1++, &fd, j2++);

    //                 // Measurement equation. The 2 PI is already multiplied on the
    //                 // LMN terms (as well as a -1 on the n).
    //                 const LmnRime lmn = s_lmns[i_comp];
    //                 SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

    //                 // Scale by envelope.
    //                 const ShapeletUV s_uv = *(s_uvs++) * one_on_lambda;
    //                 const COMPLEX envelope = get_shapelet_envelope(s_gps[i_comp], s_uv, s_num_coeffs[i_comp],
    //                                                                comps.curved_power_law_shapelet_coeffs,
    //                                                                shapelet_basis_values, sbf_l, sbf_c, chewie);

    //                 delta_vis += fd * complex * envelope;
    //             }

    //             // Visibilities are ordered over baselines and frequencies, with
    //             // baselines moving faster than frequencies.
    //             vis[i_freq * num_baselines + i_bl] += delta_vis;
    //         }
    //     }
    //     __syncthreads();
    // }

    // // List components.
    // __shared__ short s_i_freqs[NUM_THREADS_PER_BLOCK_SHAPELETS];
    // for (int i_comp_chunk = 0; i_comp_chunk < comps.num_lists * num_freqs;
    //      i_comp_chunk += NUM_THREADS_PER_BLOCK_SHAPELETS) {
    //     // Populate the shared memory.
    //     if (i_comp_chunk + threadIdx.x < comps.num_lists * num_freqs) {
    //         s_fds[threadIdx.x] = comps.list_fds[i_comp_chunk + threadIdx.x];
    //         s_gps[threadIdx.x] = comps.list_gps[i_comp_chunk + threadIdx.x];
    //         const int i_bl = (i_comp_chunk + threadIdx.x) % comps.num_lists;
    //         s_lmns[threadIdx.x] = comps.list_lmns[i_bl];
    //         s_num_coeffs[threadIdx.x] = comps.list_num_shapelet_coeffs[i_bl];
    //         s_i_freqs[threadIdx.x] = (i_comp_chunk + threadIdx.x) / comps.num_lists;
    //     } else {
    //         s_i_freqs[threadIdx.x] = -1;
    //     }
    //     __syncthreads();

    //     for (int i_bl = blockIdx.x * blockDim.x + threadIdx.x; i_bl < num_baselines; i_bl += total_num_threads) {
    //         const UVW uvw_on_hz = uvws[i_bl] / VEL_C;

    //         // Get tile indices for this baseline to get the correct beam responses.
    //         const float tile1f =
    //             floorf(-0.5f * sqrtf(4.0f * num_tiles * (num_tiles + 1.0f) - 8.0f * i_bl + 1.0f) + num_tiles + 0.5f);
    //         const int i_tile2 = i_bl - (int)(tile1f * (num_tiles - (tile1f + 1.0f) / 2.0f)) + 1;
    //         const int i_tile1 = (int)tile1f;

    //         // `i_j1_row` and `i_j2_row` are indices into beam responses.
    //         const int i_j1_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile1]];
    //         const int i_j2_row = tile_map[tile_index_to_unflagged_tile_index_map[i_tile2]];

    //         ShapeletUV *s_uvs = comps.list_shapelet_uvs + i_bl * comps.num_lists;

    //         COMPLEX complex;
    //         JONES delta_vis = JONES{
    //             .j00_re = 0.0,
    //             .j00_im = 0.0,
    //             .j01_re = 0.0,
    //             .j01_im = 0.0,
    //             .j10_re = 0.0,
    //             .j10_im = 0.0,
    //             .j11_re = 0.0,
    //             .j11_im = 0.0,
    //         };

    //         for (int i_comp = 0;
    //              i_comp + i_comp_chunk < comps.num_lists * num_freqs && i_comp < NUM_THREADS_PER_BLOCK_SHAPELETS;
    //              i_comp++) {
    //             const int i_freq = s_i_freqs[i_comp];
    //             const FLOAT freq = freqs[i_freq];
    //             const UVW uvw = uvw_on_hz * freq;

    //             const int i_col = freq_map[i_freq];
    //             const JONES *j1 = beam_jones + num_directions * (num_fee_freqs * i_j1_row + i_col);
    //             const JONES *j2 = beam_jones + num_directions * (num_fee_freqs * i_j2_row + i_col);

    //             // Get the flux density for this frequency.
    //             JONES fd = s_fds[i_comp];
    //             apply_beam(j1, &fd, j2);

    //             const LmnRime lmn = s_lmns[i_comp];
    //             SINCOS(uvw.u * lmn.l + uvw.v * lmn.m + uvw.w * lmn.n, &complex.y, &complex.x);

    //             // Scale by envelope.
    //             const ShapeletUV s_uv = *(s_uvs++) * (freq / VEL_C);
    //             const COMPLEX envelope =
    //                 get_shapelet_envelope(s_gps[i_comp], s_uv, s_num_coeffs[i_comp], comps.list_shapelet_coeffs,
    //                                       shapelet_basis_values, sbf_l, sbf_c, chewie);

    //             delta_vis += fd * complex * envelope;

    //             // Write the results if we're done with this frequency.
    //             if (i_comp + 1 == NUM_THREADS_PER_BLOCK_SHAPELETS || s_i_freqs[i_comp + 1] != i_freq) {
    //                 vis[i_freq * num_baselines + i_bl] += delta_vis;
    //                 delta_vis = JONES{
    //                     .j00_re = 0.0,
    //                     .j00_im = 0.0,
    //                     .j01_re = 0.0,
    //                     .j01_im = 0.0,
    //                     .j10_re = 0.0,
    //                     .j10_im = 0.0,
    //                     .j11_re = 0.0,
    //                     .j11_im = 0.0,
    //                 };
    //             }
    //         }
    //     }
    //     __syncthreads();
    // }
}

extern "C" const char *model_points(const Points *comps, const Addresses *a, const UVW *d_uvws,
                                    const JONES *d_beam_jones) {
    dim3 gridDim, blockDim;
    // Thread blocks are distributed by cross-correlation baseline.
    blockDim.x = NUM_THREADS_PER_BLOCK_POINTS;
    gridDim.x = (int)ceil((double)a->num_baselines / (double)blockDim.x);

    // #ifdef SINGLE
    // #else
    // cudaCheck(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
    // #endif
    // cudaCheck(hipFuncSetCacheConfig(reinterpret_cast<const void*>(model_points_fee_kernel_large_source_count), hipFuncCachePreferL1));

    // if (comps->num_power_laws + comps->num_curved_power_laws + comps->num_lists > 1000000) {
    //     model_points_fee_kernel_large_source_count<<<gridDim, blockDim>>>(
    //         a->num_freqs, a->num_baselines, a->d_freqs, d_uvws, *comps, (JONES *)d_beam_jones, a->d_tile_map,
    //         a->d_freq_map, a->num_unique_beam_freqs, a->d_tile_index_to_unflagged_tile_index_map, a->d_vis);
    // } else {
    //     blockDim.x = NUM_THREADS_PER_BLOCK_POINTS;
    gridDim.x = (int)ceil((double)(a->num_baselines * a->num_freqs) / (double)blockDim.x);
    model_points_fee_kernel_small_source_count<<<gridDim, blockDim>>>(
        a->num_freqs, a->num_baselines, a->d_freqs, d_uvws, *comps, d_beam_jones, a->d_tile_map, a->d_freq_map,
        a->num_unique_beam_freqs, a->d_tile_index_to_unflagged_tile_index_map, a->d_vis);
    // }

    hipError_t error_id = hipDeviceSynchronize();
    if (error_id != hipSuccess) {
        return hipGetErrorString(error_id);
    }
    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        return hipGetErrorString(error_id);
    }

    return NULL;
}

extern "C" const char *model_gaussians(const Gaussians *comps, const Addresses *a, const UVW *d_uvws,
                                       const JONES *d_beam_jones) {
    dim3 gridDim, blockDim;
    // Thread blocks are distributed by cross-correlation baseline.
    blockDim.x = NUM_THREADS_PER_BLOCK_GAUSSIANS;
    gridDim.x = (int)ceil((double)a->num_baselines / (double)blockDim.x);

    model_gaussians_fee_kernel_large_source_count<<<gridDim, blockDim>>>(
        a->num_freqs, a->num_baselines, a->d_freqs, d_uvws, *comps, d_beam_jones, a->d_tile_map, a->d_freq_map,
        a->num_unique_beam_freqs, a->d_tile_index_to_unflagged_tile_index_map, a->d_vis);

    hipError_t error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        return hipGetErrorString(error_id);
    }

    return NULL;
}

extern "C" const char *model_shapelets(const Shapelets *comps, const Addresses *a, const UVW *d_uvws,
                                       const JONES *d_beam_jones) {
    dim3 gridDim, blockDim;
    // Thread blocks are distributed by cross-correlation baseline.
    blockDim.x = NUM_THREADS_PER_BLOCK_SHAPELETS;
    gridDim.x = (int)ceil((double)a->num_baselines / (double)blockDim.x);

    model_shapelets_fee_kernel_large_source_count<<<gridDim, blockDim>>>(
        a->num_freqs, a->num_baselines, a->d_freqs, d_uvws, *comps, a->d_shapelet_basis_values, a->sbf_l, a->sbf_c,
        a->sbf_dx, d_beam_jones, a->d_tile_map, a->d_freq_map, a->num_unique_beam_freqs,
        a->d_tile_index_to_unflagged_tile_index_map, a->d_vis);

    hipError_t error_id = hipGetLastError();
    if (error_id != hipSuccess) {
        return hipGetErrorString(error_id);
    }

    return NULL;
}
