// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

// "Homegrown" CUDA utilities.
//
// As this code contains code derived from an official NVIDIA example
// (https://github.com/NVIDIA/cuda-samples/blob/master/Samples/1_Utilities/deviceQuery/deviceQuery.cpp),
// legally, a copyright, list of conditions and disclaimer must be distributed
// with this code. This should be found in the "cuda" directory of the
// mwa_hyperdrive git repo, file LICENSE-NVIDIA.

#include <hip/hip_runtime.h>

extern "C" const char *get_cuda_error(int cuda_error_id) {
    hipError_t error_id = static_cast<hipError_t>(cuda_error_id);
    return hipGetErrorString(error_id);
}

extern "C" int get_cuda_device_info(int device, char name[256], int *device_major, int *device_minor,
                                    size_t *total_global_mem, int *driver_version, int *runtime_version) {
    hipError_t error_id = hipSetDevice(device);
    if (error_id != hipSuccess)
        // Can't pass `cudaError_t` over the FFI boundary. Cast to an int.
        return static_cast<int>(error_id);

    hipDeviceProp_t device_prop;
    error_id = hipGetDeviceProperties(&device_prop, device);
    if (error_id != hipSuccess)
        return static_cast<int>(error_id);

    memcpy(name, device_prop.name, 256);
    *device_major = device_prop.major;
    *device_minor = device_prop.minor;
    *total_global_mem = device_prop.totalGlobalMem;

    error_id = hipDriverGetVersion(driver_version);
    if (error_id != hipSuccess)
        return static_cast<int>(error_id);

    error_id = hipRuntimeGetVersion(runtime_version);
    if (error_id != hipSuccess)
        return static_cast<int>(error_id);

    return hipError_t::hipSuccess;
}
