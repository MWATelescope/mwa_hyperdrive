// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#include "common.cuh"
#include "memory.h"

Addresses init_model(const size_t num_baselines, const size_t num_freqs, const size_t sbf_l, const size_t sbf_n,
                     const FLOAT sbf_c, const FLOAT sbf_dx, const UVW *uvws, const FLOAT *freqs,
                     const FLOAT *shapelet_basis_values, JonesF32 *vis) {
    UVW *d_uvws = NULL;
    size_t size_uvws = num_baselines * sizeof(UVW);
    cudaSoftCheck(hipMalloc(&d_uvws, size_uvws));
    cudaSoftCheck(hipMemcpy(d_uvws, uvws, size_uvws, hipMemcpyHostToDevice));

    FLOAT *d_freqs = NULL;
    size_t size_freqs = num_freqs * sizeof(FLOAT);
    cudaSoftCheck(hipMalloc(&d_freqs, size_freqs));
    cudaSoftCheck(hipMemcpy(d_freqs, freqs, size_freqs, hipMemcpyHostToDevice));

    FLOAT *d_shapelet_basis_values = NULL;
    size_t size_sbfs = sbf_l * sbf_n * sizeof(FLOAT);
    cudaSoftCheck(hipMalloc(&d_shapelet_basis_values, size_sbfs));
    cudaSoftCheck(hipMemcpy(d_shapelet_basis_values, shapelet_basis_values, size_sbfs, hipMemcpyHostToDevice));

    JonesF32 *d_vis = NULL;
    size_t num_vis = num_baselines * num_freqs;
    size_t size_vis = num_vis * sizeof(JonesF32);
    cudaSoftCheck(hipMalloc(&d_vis, size_vis));
    cudaSoftCheck(hipMemcpy(d_vis, vis, size_vis, hipMemcpyHostToDevice));

    return Addresses{.num_freqs = num_freqs,
                     .num_vis = num_vis,
                     .sbf_l = sbf_l,
                     .sbf_n = sbf_n,
                     .sbf_c = sbf_c,
                     .sbf_dx = sbf_dx,
                     .d_uvws = d_uvws,
                     .d_freqs = d_freqs,
                     .d_shapelet_basis_values = d_shapelet_basis_values,
                     .d_vis = d_vis,
                     .host_vis = vis};
}

void copy_vis(const Addresses *a) {
    cudaSoftCheck(hipMemcpy(a->host_vis, a->d_vis, a->num_vis * sizeof(JonesF32), hipMemcpyDeviceToHost));
}

void clear_vis(const Addresses *a) { hipMemset(a->d_vis, 0.0, a->num_vis * sizeof(JonesF32)); }

void destroy(const Addresses *a) {
    cudaSoftCheck(hipFree(a->d_uvws));
    cudaSoftCheck(hipFree(a->d_freqs));
    cudaSoftCheck(hipFree(a->d_shapelet_basis_values));
    cudaSoftCheck(hipFree(a->d_vis));
}
