#include "hip/hip_runtime.h"
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

// Derived from WODEN (https://github.com/JLBLine/WODEN, commit 854d9c8).

#include <assert.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#include "hyperdrive.h"

// `gpu_assert` checks that CUDA code successfully returned.
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "gpu_assert: %s (%s:%d)\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}
// `cudaCheck` wraps `gpu_assert` for general usage.
#define cudaCheck(code)                                                                                                \
    { gpu_assert((code), __FILE__, __LINE__); }

#ifndef NDEBUG
// When not debugging, `cudaSoftCheck` is a "no-op". Useful for granting full speed in release builds.
#define cudaSoftCheck(code) (code)
#else
#define cudaSoftCheck(code)                                                                                            \
    { gpu_assert((code), __FILE__, __LINE__); }
#endif

/// Kernel for calculating point-source visibilities. Uses `atomicAdd` to fold
/// the values over source components.
__global__ void calc_point_vis(const unsigned int n_baselines, const unsigned int n_points, const unsigned int n_vis,
                               const float *d_u, const float *d_v, const float *d_w, const float *d_l, const float *d_m,
                               const float *d_n, const float *d_point_fd, float *d_sum_vis_real,
                               float *d_sum_vis_imag) {
    const int i_vis = threadIdx.x + (blockDim.x * blockIdx.x);
    const int i_comp = threadIdx.y + (blockDim.y * blockIdx.y);

    if ((i_vis >= n_vis) || (i_comp >= n_points))
        return;

    // All present frequencies change every `n_baselines`.
    const int i_freq = i_vis / n_baselines;

    const float u = d_u[i_vis];
    const float v = d_v[i_vis];
    const float w = d_w[i_vis];
    const float l = d_l[i_comp];
    const float m = d_m[i_comp];
    const float n = d_n[i_comp];
    const float flux_density = d_point_fd[i_freq * n_points + i_comp];

    // Calculate -2 * PI * (u * l + v * m + w * (n - 1)). We don't use PI
    // explicitly; CUDA's sincospif does that.
    // Not sure why, but we get an exact match with OSKAR sims and correct
    // location on sky through wsclean without negative in front of 2pi.
    float real;
    float imag;
    sincospif(2 * (u * l + v * m + w * (n - 1.0f)), &imag, &real);

    atomicAdd(&d_sum_vis_real[i_vis], real * flux_density);
    atomicAdd(&d_sum_vis_imag[i_vis], imag * flux_density);
}

/// Generate visibilities for the given source list.
///
/// Currently only takes a single source containing point sources.
extern "C" int vis_gen(const UVW_s *uvw, const Source_s *src, Visibilities_s *vis) {
    // Sanity checks.
    // TODO: Check other source component types when they are being handled.
    if (src->n_points == 0) {
        fprintf(stderr, "No point sources provided; nothing to do!\n");
        return 1;
    }

    float *d_u = NULL;
    float *d_v = NULL;
    float *d_w = NULL;
    size_t size_baselines = uvw->n_vis * sizeof(float);
    cudaSoftCheck(hipMalloc(&d_u, size_baselines));
    cudaSoftCheck(hipMalloc(&d_v, size_baselines));
    cudaSoftCheck(hipMalloc(&d_w, size_baselines));
    cudaSoftCheck(hipMemcpy(d_u, uvw->u, size_baselines, hipMemcpyHostToDevice));
    cudaSoftCheck(hipMemcpy(d_v, uvw->v, size_baselines, hipMemcpyHostToDevice));
    cudaSoftCheck(hipMemcpy(d_w, uvw->w, size_baselines, hipMemcpyHostToDevice));

    float *d_l = NULL;
    float *d_m = NULL;
    float *d_n = NULL;
    size_t size_points = src->n_points * sizeof(float);
    cudaSoftCheck(hipMalloc(&d_l, size_points));
    cudaSoftCheck(hipMalloc(&d_m, size_points));
    cudaSoftCheck(hipMalloc(&d_n, size_points));
    cudaSoftCheck(hipMemcpy(d_l, src->point_l, size_points, hipMemcpyHostToDevice));
    cudaSoftCheck(hipMemcpy(d_m, src->point_m, size_points, hipMemcpyHostToDevice));
    cudaSoftCheck(hipMemcpy(d_n, src->point_n, size_points, hipMemcpyHostToDevice));

    float *d_point_fd = NULL;
    size_t size_fds = size_points * src->n_channels;
    cudaSoftCheck(hipMalloc(&d_point_fd, size_fds));
    cudaSoftCheck(hipMemcpy(d_point_fd, src->point_fd, size_fds, hipMemcpyHostToDevice));

    float *d_sum_vis_real = NULL;
    float *d_sum_vis_imag = NULL;
    size_t size_visibilities = vis->n_visibilities * sizeof(float);
    cudaSoftCheck(hipMalloc(&d_sum_vis_real, size_visibilities));
    cudaSoftCheck(hipMalloc(&d_sum_vis_imag, size_visibilities));

    dim3 blocks, threads;

    // Generate visibilities for the point sources.
    if (src->n_points > 0) {
        // Thread blocks are distributed by visibility (one visibility per
        // frequency and baseline; y) and point source component (y);
        threads.x = 64;
        threads.y = 16;
        blocks.x = (int)ceilf((float)uvw->n_vis / (float)threads.x);
        blocks.y = (int)ceilf((float)src->n_points / (float)threads.y);

#ifndef NDEBUG
        printf("num. visibilities (uvw->n_elem): %u\n", uvw->n_vis);
        printf("num. point source components (src->n_points): %u\n", src->n_points);
        printf("num. x blocks (uvw->n_elem): %u\n", blocks.x);
        printf("num. y blocks (src->n_points): %u\n", blocks.y);
#endif

        calc_point_vis<<<blocks, threads>>>(uvw->n_baselines, src->n_points, uvw->n_vis, d_u, d_v, d_w, d_l, d_m, d_n,
                                            d_point_fd, d_sum_vis_real, d_sum_vis_imag);
        cudaCheck(hipPeekAtLastError());
        cudaSoftCheck(hipFree(d_point_fd));
    } // if (num_points > 0)

    // Copy the results into host memory.
    cudaSoftCheck(hipMemcpy(vis->real, d_sum_vis_real, size_visibilities, hipMemcpyDeviceToHost));
    cudaSoftCheck(hipMemcpy(vis->imag, d_sum_vis_imag, size_visibilities, hipMemcpyDeviceToHost));

    // Clean up.
    cudaSoftCheck(hipFree(d_u));
    cudaSoftCheck(hipFree(d_v));
    cudaSoftCheck(hipFree(d_w));
    cudaSoftCheck(hipFree(d_l));
    cudaSoftCheck(hipFree(d_m));
    cudaSoftCheck(hipFree(d_n));
    cudaSoftCheck(hipFree(d_sum_vis_real));
    cudaSoftCheck(hipFree(d_sum_vis_imag));

    return 0;
}
